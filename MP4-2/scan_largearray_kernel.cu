#include "hip/hip_runtime.h"
extern const int NUM_BANKS = 16;
extern const int LOG_NUM_BANKS = 4;
extern const int TILE_SIZE = 1024;
// You can use any other block size you wish.
extern const int BLOCK_SIZE = (TILE_SIZE / 2);



// Device Functions






// Kernel Functions

__global void add_all(unsigned int *data, int n, unsigned int *aux)
{
	__shared__ unsigned int b = aux[blockIdx.x];
	unsigned int t = threadIdx.x + blockDim.x * blockIdx.x;

	if (t < n) {
		int a = data[t];
		a += b;
		data[t] = a;
	}
}
		

__global__ void scan_kernel(unsigned int *in_data, unsigned int *out_data, int n, unsigned int *aux)
{
	//setup + readin
	__shared__ scan_array[BLOCK_SIZE * 2];

	unsigned int t = threadIdx.x;
	unsigned int start = 2 * blockDim.x * blockIdx.x;
	scan_array[t] = start + 2 >= n ? 0 : in_data[start + t];
	scan_array[blockDim.x + t] = start + blockDim.x + t >= n ? 0 : in_data[start + blockDim.x + t];

	//reduction step
	int stride = 1;
	while (stride <= BLOCK_SIZE) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index < 2 * BLOCK_SIZE) {
			scan_array[index] += scan_array[index - stride];
		}
		stride <<= 1;
		__syncthreads();
	}
	
	//distribution step
	stride = BLOCK_SIZE / 2;
	while (stride) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if ((index + stride) < 2 * BLOCK_SIZE) {
			scan_array[index + stride] += scan_array[index];
		}
		
		stride >>= 1;
		__syncthreads();
	}

	//writeout
	if (start + t < n) {
		out_data[start + t] = scan_array[t];
	}

	if (start + blockDim.x + t < n) {
		out_data[start + blockDim.x + t] = scan_array[t + blockDim.x];
	}

	if (t == 0) {
		aux[blockIdx.x] = scan_array[2 * BLOCK_SIZE - 1];
	}

}


// Host Helper Functions (allocate your own data structure...)

// **===-------- Modify the body of this function -----------===**
// You may need to make multiple kernel calls. Make your own kernel
// functions in this file, and then call them from here.
// Note that the code has been modified to ensure numElements is a multiple 
// of TILE_SIZE
void prescanArray(unsigned int *outArray, unsigned int *inArray, unsigned int* blockSums, int numElements);

void prescanArray(unsigned int *outArray, unsigned int *inArray, unsigned int* blockSums, int numElements)
{   //all pointers are to device memory regions
    int tiles = (numElements + TILE_SIZE - 1) / TILE_SIZE;
    scan_kernel<<<tiles, BLOCK_SIZE>>>(inArray, outArray, numElements, blockSums);

    if (tiles > 1) {
	    unsigned int *nextBlocks = preallocBlockSums(tiles);
	    prescanArray(blockSums, blockSums, nextBlocks, tiles);
	    deallocBlockSums(nextBlocks);
    }

    add_all<<<tiles - 1, BLOCK_SIZE>>>(outArray + TILE_SIZE, numElements - TILE_SIZE, blockSums);

}
// **===-----------------------------------------------------------===**

// Use the function to allocate your block sums here
unsigned int* preallocBlockSums(int num_elements)
{
	unsigned int* bs_d = 0; //assign your device memory pointer to this variable
	// =========================================

	int blocks = (num_elements + TILE_SIZE - 1) / TILE_SIZE;
	hipMalloc((void **) &bs_d, blocks * sizeof(unsigned int));

	// =========================================
	return bs_d;
}

// Use the function to deallocate (free) your block sums
void deallocBlockSums(unsigned int* bs_d)
{
	hipFree(bs_d);
}
